#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#include <math.h>

extern "C" 
{
	__device__ float sim_inv_cdf( float mu, float sigma, float phi_a, float phi_b, hiprandState localState)
	{	//Generate trunc StdNorm by inverse CDF if conditions are stable.
		float u = ((phi_b - phi_a) * hiprand_uniform(&localState)) + phi_a;
		float x = sigma * normcdfinvf(u) + mu;
		return x;
	} //Gets valid sample every time--so speed, very wow. 

	__device__ float simple_rejection( float mu, float sigma, float stdA, float stdB, int maxtries, hiprandState localState)
	{
		float res = 1.0f/0.0f; //set result to inf. For error checking.
		float z;

		for(int i=0; i < maxtries; i++)
		{
			z = hiprand_normal(&localState);
			if ( z <= stdB && z >= stdA) 
			{
				res = mu + (sigma*z);
				return res;
			}
		} //end maxtry for loop
		return res;
	} //end simple rejection-sampler.

	__device__ float one_sided_trunc( float mu, float sigma, float stdA, float stdB, hiprandState localState)
	{
		float logRho, z, logU, res, alpha, trunc;
		
		if (isinf(stdA)) trunc=-stdB; else trunc=stdA; //If a is -Inf, then b is truncated. Else, a is truncated. 

		alpha = (trunc + sqrtf(trunc*trunc + 4))/2; //Optimal alpha
		do { //Start rejection sample loop.
			z = (-logf( hiprand_uniform(&localState) )/ alpha) + trunc; //Truncated Exponential
			if (trunc < alpha) logRho = -((z-alpha) * (z-alpha)) / 2;
			else logRho = ((trunc-alpha)*(trunc-alpha) -(alpha-z)*(alpha-z)) / 2;
			logU = logf(hiprand_uniform(&localState));
		} while (logU > logRho);
		
		//If left trunc, do as usual. If right trunc, reflect the z value, then add mu.
		if (isinf(stdB)) res = mu + sigma * z;
		else res = mu - sigma*z;
		return res;
	} //end one-sided sampler.

	__device__ float robert( float mu, float sigma, float stdA, float stdB, hiprandState localState)
	{	//Condns unstable for inv-CDF. Do Robert (2009).
		float logrho, z, logu, res;
		do {
			z = (stdB-stdA) * hiprand_uniform(&localState) + stdA;
			logu = logf( hiprand_uniform(&localState) );
			if ( stdA<=0 && stdB>= 0 ) logrho = -(z*z)/2;
			else if (stdA > 0) logrho = -((stdA*stdA)-(z*z))/2;
			else logrho = -((stdB*stdB)-(z*z))/2;
		} while(logu > logrho); 

		res = sigma*z + mu;
		return res;
	} //end Robert sampler.
	
	__global__ void truncnormal_kernel(float *result, int n, float *mu, 
						float *sigma, float *a, float *b, int maxtries,
						int mu_len, int sigma_len, int a_len, int b_len)
	{
		int myblock = blockIdx.x + blockIdx.y * gridDim.x;
		int blocksize = blockDim.x * blockDim.y * blockDim.z;
		int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
		int idx = myblock * blocksize + subthread;

		if (idx >= n) return; //Index is larger than sample size--do no calculation here.
		hiprandState localState;
		hiprand_init(idx, idx, 0, &localState);

		//Declare vars in thread-local memory.
		float t_a = a[idx % a_len];
		float t_b = b[idx % b_len];
		float t_mu = mu[idx % mu_len];
		float t_sigma = sigma[idx % sigma_len];
		float res = 1.0f/0.0f;
		
		float stdA = (t_a - t_mu)/t_sigma; //Standardize truncation points. Done in-thread.
		float stdB = (t_b - t_mu)/t_sigma;
		
		float phi_a = normcdff(stdA); //Calculate CDF of trunc points from StdNormal. 
		float phi_b = normcdff(stdB);

		if ( phi_b - phi_a > 0.02f )
		{ //If stable conditions, use inverse-CDF.
			result[idx] = sim_inv_cdf(t_mu, t_sigma, phi_a, phi_b, localState); 
			return;
		}

		else if ( isinf(stdA) || isinf(stdB) ) 
		{ //One-sided truncation. 
			res = one_sided_trunc( t_mu, t_sigma, stdA, stdB, localState);
			result[idx] = res;
			return;
		}

		else if ( stdB-stdA >= sqrtf(6.2831853f) && phi_b-phi_a > 0.0001f)
		{ //Two-sided trunc with truncs far away--do the naive rejection sampler
			res = simple_rejection(t_mu, t_sigma, stdA, stdB, maxtries, localState);
			if (!isinf(res))
			{
				result[idx] = res;
				return;
			}
		}
		else{ //Do the Robert method.
			res = robert(t_mu, t_sigma, stdA, stdB, localState); 
			result[idx] = res;
			return;
		} //End truncation regions on same side of mean.
	} //end truncnorm kernel
} //end extern C.
